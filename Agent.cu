#include "hip/hip_runtime.h"




#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
//#include "qlearning.h"
//#include "draw_env.h"
//#include "common_def.h"
#include <random>
#include <time.h>


static hiprandState *states = NULL;
int N = 512;
//int nx_1 = 32;
//int ny_1 = 16;
//dim3 block_1(nx_1, ny_1);
//dim3 grid_1((nx_1 + block_1.x - 1) / block_1.x, (ny_1 + block_1.y - 1) / block_1.y);

// Kernel functions

__global__ void k_Agent_init(hiprandState *states, float* q_table,  int nx, int ny)
{
	//printf("Hello from init\n");
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int tid = iy * nx + ix;
	//unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState *state = states + 0;
	hiprand_init(clock() + tid, tid, 0, state);
	if (tid < nx * ny)
	{
		q_table[tid] = hiprand_uniform(state) * 0.1;
	}


}

__global__ void k_Agent_update_Qtable(short* action, float* Q_table, float gamma, int2* curr_state, int2* next_state, float* reward, int* flag_action)
{
	// printf("Hello from Q update\n");
	//printf("Hello World from update!\n");
	float maxQ;
	float alpha = 0.8;
	int c_index, index;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int local_flag = flag_action[tid];
	int2 currs = curr_state[tid];
	int2 nexts = next_state[tid];
	float reward_local = reward[tid];
	short local_action = action[tid];
	if (local_flag != 0 && tid < 512)
	{
		//printf("reward = %d\n", local_flag);
		if (reward_local == 0) {
			c_index = (nexts.y * 46 + nexts.x) * 4 + 0;
			maxQ = Q_table[c_index];
			for (int i = 0; i < 4; i++)
			{
				if (Q_table[c_index + i] > maxQ)
				{
					maxQ = Q_table[c_index + i];
				}
			}
		}
		else
		{
			maxQ = 0;
		}
		if(reward_local == 1 || reward_local == -1 || reward_local == 0)
		Q_table[((currs.y) * 46 + currs.x) * 4 + local_action] += alpha * (reward_local + gamma * maxQ - Q_table[((currs.y) * 46 + currs.x) * 4 + local_action]);
		
		if (reward_local != 0)
		{
			flag_action[tid] = 0;
			//printf("reward = %f\n", reward_local);
		}
		//if (reward_local != 1.0 && reward_local != -1.0 && reward_local != 0.0) printf("reward = %f\n", reward_local);
		
	}
	


}



__global__ void k_Agent_adjustepselon(float* k_epsilon)
{

	k_epsilon[0] = k_epsilon[0] - 0.005;
}


__global__ void k_Agent_action(hiprandState *states, float* Q_table, int2* cstate, short* action, float epselon)
{
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//__shared__ int2 s[512];
	//float Q = Q_table[tid];
	//float Q_new;
	//unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	//unsigned int iy = threadIdx.x + blockIdx.x * blockDim.x;
	//unsigned int tid = iy * 32 + ix;
	int2 currs = cstate[tid];
	float maxQ = -10000;
	
	hiprandState *state = states + 0;
	float dOut;

	hiprand_init(clock() + tid, tid, 0, state);
	dOut = hiprand_uniform(state);
	
	
	//s[tid] = cstate[tid]; 
	//__syncthreads();
	
		if (dOut < epselon)
		{

			action[tid] = (short)(hiprand_uniform(state) * 4);


		}
		else
		{


			//maxQ = Q_table[((currs.y) * 46 + currs.x) * 4];
			//action[tid] = 0;
			for (int i = 0; i < 4; i++)
			{
				if (maxQ < Q_table[((currs.y) * 46 + currs.x) * 4 + i])
				{
					maxQ = Q_table[((currs.y) * 46 + currs.x) * 4 + i];
					action[tid] = (short)i;
				}

			}

		}
	
}

	



__global__ void k_Agent_clearaction(int* flag_agent, short* action)
{
	//printf("Hello from clear\n");
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 512)
	{
		flag_agent[tid] = 1;
		action[tid] = 0;
	}

	
}




class Agent
{
public:

	float* q_Table;
	float* k_epsilon;
	float epselon = 1.0;
	float gamma = 0.7;
	//int count_alive_agent;
	int *flag_alive;
	short* action;
	int nx = 4;
	int ny = 46 * 46;
	int nBytes = (nx * ny) * sizeof(float);

	Agent();

	~Agent();

	// Mutator
	void Agent_init();
	void Agent_adjustepselon();
	void Agent_update_Qtable(int2* curr_state, int2* next_state, float* reward);
	void Agent_action(int2* cstate);
	void Agent_clearaction();
};


// Create member functions
Agent::Agent()
{

}
Agent::~Agent()
{

}

//Member functions
void Agent::Agent_init()
{
	hipMalloc((void **)&k_epsilon, 1 * sizeof(float));
	hipMalloc((void **)&action,  sizeof(short)* N);
	hipMalloc((void **)&q_Table, nBytes);
	hipMalloc((void **)&states, sizeof(hiprandState) * 1 * 1);
	hipMalloc((void **)&flag_alive, sizeof(int) * N);
	dim3 block(nx, ny);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
	k_Agent_init << <grid, block >> > (states, q_Table, nx, ny);
	k_Agent_clearaction << <8, 64 >> > (flag_alive, action);
}


void Agent::Agent_adjustepselon()
{
	//hipMemcpy(k_epsilon, &(epselon), sizeof(float), hipMemcpyHostToDevice);
	//k_Agent_adjustepselon << < 1, 1 >> > (k_epsilon);
	//hipMemcpy(&epselon, k_epsilon, sizeof(float), hipMemcpyDeviceToHost);
	epselon = epselon - 0.005;
	if (epselon < 0.0) epselon = 0.0;
}


void Agent::Agent_update_Qtable(int2* curr_state, int2* next_state, float* rewards)
{

	k_Agent_update_Qtable << <8, 64>> > (action, q_Table, gamma, curr_state, next_state, rewards, flag_alive);

}



void Agent::Agent_action(int2* cstate)
{
	
	k_Agent_action << < 8, 64 >> > (states, q_Table, cstate, action, epselon);
}

void Agent::Agent_clearaction()
{
	k_Agent_clearaction << <8, 64 >> > (flag_alive, action);
}


Agent my_agent;


//Interface functions
void agent_init()
{
	my_agent.Agent_init();
}

void agent_clearaction()
{
	my_agent.Agent_clearaction();
}

float agent_adjustepsilon()
{
	my_agent.Agent_adjustepselon();
	return my_agent.epselon;
}

void agent_update(int2* cstate, int2* nstate, float* rewards)
{
	my_agent.Agent_update_Qtable(cstate, nstate, rewards);
}

short* agent_action(int2* cstate)
{

	my_agent.Agent_action(cstate);

	return my_agent.action;
}

